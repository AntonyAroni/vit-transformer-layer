
#include <hip/hip_runtime.h>
#ifdef USE_CUDA
#include "../../include/cuda/cuda_matrix.h"
#include <iostream>

cublasHandle_t CudaMatrix::cublas_handle;

CudaMatrix::CudaMatrix(int rows, int cols) : rows(rows), cols(cols) {
    cudaMalloc(&d_data, rows * cols * sizeof(float));
}

CudaMatrix::CudaMatrix(const Matrix& host_matrix) : rows(host_matrix.getRows()), cols(host_matrix.getCols()) {
    cudaMalloc(&d_data, rows * cols * sizeof(float));
    copyFromHost(host_matrix);
}

CudaMatrix::~CudaMatrix() {
    cudaFree(d_data);
}

void CudaMatrix::copyFromHost(const Matrix& host_matrix) {
    float* temp = new float[rows * cols];
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            temp[i * cols + j] = (float)host_matrix(i, j);
        }
    }
    cudaMemcpy(d_data, temp, rows * cols * sizeof(float), cudaMemcpyHostToDevice);
    delete[] temp;
}

void CudaMatrix::copyToHost(Matrix& host_matrix) {
    float* temp = new float[rows * cols];
    cudaMemcpy(temp, d_data, rows * cols * sizeof(float), cudaMemcpyDeviceToHost);
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            host_matrix(i, j) = (double)temp[i * cols + j];
        }
    }
    delete[] temp;
}

CudaMatrix CudaMatrix::matmul(const CudaMatrix& a, const CudaMatrix& b) {
    CudaMatrix result(a.rows, b.cols);
    const float alpha = 1.0f, beta = 0.0f;
    
    cublasSgemm(cublas_handle, CUBLAS_OP_N, CUBLAS_OP_N,
                b.cols, a.rows, a.cols,
                &alpha, b.d_data, b.cols,
                a.d_data, a.cols,
                &beta, result.d_data, b.cols);
    
    return result;
}

void CudaMatrix::initCublas() {
    cublasCreate(&cublas_handle);
}

void CudaMatrix::destroyCublas() {
    cublasDestroy(cublas_handle);
}
#endif