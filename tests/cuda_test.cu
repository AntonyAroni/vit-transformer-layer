#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>

__global__ void vectorAdd(float *a, float *b, float *c, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    std::cout << "=== CUDA TEST ===" << std::endl;
    
    // Check CUDA devices
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    std::cout << "CUDA devices found: " << deviceCount << std::endl;
    
    if (deviceCount == 0) {
        std::cout << "❌ No CUDA devices found!" << std::endl;
        return -1;
    }
    
    // Get device properties
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    std::cout << "Device 0: " << prop.name << std::endl;
    std::cout << "Compute capability: " << prop.major << "." << prop.minor << std::endl;
    std::cout << "Global memory: " << prop.totalGlobalMem / (1024*1024) << " MB" << std::endl;
    
    // Test basic CUDA kernel
    const int N = 1024;
    float *h_a, *h_b, *h_c;
    float *d_a, *d_b, *d_c;
    
    // Allocate host memory
    h_a = new float[N];
    h_b = new float[N];
    h_c = new float[N];
    
    // Initialize arrays
    for (int i = 0; i < N; i++) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }
    
    // Allocate device memory
    hipMalloc(&d_a, N * sizeof(float));
    hipMalloc(&d_b, N * sizeof(float));
    hipMalloc(&d_c, N * sizeof(float));
    
    // Copy to device
    hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice);
    
    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    
    std::cout << "\nLaunching CUDA kernel..." << std::endl;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N);
    
    // Check for errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cout << "❌ CUDA kernel error: " << hipGetErrorString(err) << std::endl;
        return -1;
    }
    
    // Copy result back
    hipMemcpy(h_c, d_c, N * sizeof(float), hipMemcpyDeviceToHost);
    
    // Verify results
    bool success = true;
    for (int i = 0; i < 10; i++) {
        float expected = h_a[i] + h_b[i];
        if (abs(h_c[i] - expected) > 1e-5) {
            success = false;
            break;
        }
    }
    
    if (success) {
        std::cout << "✅ CUDA kernel test passed!" << std::endl;
    } else {
        std::cout << "❌ CUDA kernel test failed!" << std::endl;
    }
    
    // Test cuBLAS
    std::cout << "\nTesting cuBLAS..." << std::endl;
    hipblasHandle_t handle;
    hipblasStatus_t stat = hipblasCreate(&handle);
    
    if (stat == HIPBLAS_STATUS_SUCCESS) {
        std::cout << "✅ cuBLAS initialized successfully!" << std::endl;
        
        // Simple matrix multiplication test
        const int M = 64, N = 64, K = 64;
        float *d_A, *d_B, *d_C;
        
        hipMalloc(&d_A, M * K * sizeof(float));
        hipMalloc(&d_B, K * N * sizeof(float));
        hipMalloc(&d_C, M * N * sizeof(float));
        
        // Initialize with ones
        float *h_ones = new float[M * K];
        for (int i = 0; i < M * K; i++) h_ones[i] = 1.0f;
        
        hipMemcpy(d_A, h_ones, M * K * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_B, h_ones, K * N * sizeof(float), hipMemcpyHostToDevice);
        
        const float alpha = 1.0f, beta = 0.0f;
        stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                          N, M, K, &alpha,
                          d_B, N, d_A, K,
                          &beta, d_C, N);
        
        if (stat == HIPBLAS_STATUS_SUCCESS) {
            std::cout << "✅ cuBLAS matrix multiplication successful!" << std::endl;
        } else {
            std::cout << "❌ cuBLAS matrix multiplication failed!" << std::endl;
        }
        
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        delete[] h_ones;
        
        hipblasDestroy(handle);
    } else {
        std::cout << "❌ cuBLAS initialization failed!" << std::endl;
    }
    
    // Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    delete[] h_a;
    delete[] h_b;
    delete[] h_c;
    
    std::cout << "\n✅ CUDA test completed!" << std::endl;
    
    return 0;
}